/*
 * Driver program for a CUDA-based A5/1 rainbow table generator.
 *
 * Copyright (C) 2009: Ingo Albrecht <prom@berlin.ccc.de>
 */

#ifndef TEST_INTERMEDIATES
/* values below are for normal runs */

/*
 * These values are appropriate for a Quadro FX 570M.
 *
 * Before running this on different hardware, you
 * should decrease OPERATIONS_PER_RUN and then
 * increase it incrementally until you get
 * run lengths approaching 5 seconds.
 *
 * Thread and block count should be selected
 * so that they almost hit the register bound.
 *
 * If you want to tune the code for your card,
 * you should do it incrementally, keeping
 * the run length below 5 seconds, or your
 * graphics subsystem might go wonky.
 */

// number of threads per block
#define NUM_THREADS 32

// number of blocks to schedule
#define NUM_BLOCKS  32

// how long each run should be in cycles.
// must be a power of two for now.
#define OPERATIONS_PER_RUN  32768

#else
// values below are for intermediate testing

#define NUM_THREADS 10
#define NUM_BLOCKS  1

#define OPERATIONS_PER_RUN 32768

#endif

// total operations per chain (2^21)
#define OPERATIONS_PER_CHAIN 2097152

// number of chains to be computed
#define NUM_CHAINS NUM_THREADS * NUM_BLOCKS


#include <stdio.h>
#include <unistd.h>

#include <cutil.h>

#include "calculate_chain_kernel.cu"

int
main(int argc, char **argv) {
  CUT_DEVICE_INIT(argc, argv);

  uint32 i;

  uint64 start = 0; // XXX put your start vector here
  
  printf("Computing %d chains divided into %d blocks of %d threads, starting at 0x%16.16llx\n",
         NUM_CHAINS, NUM_BLOCKS, NUM_THREADS, start);

  uint32 num_runs = OPERATIONS_PER_CHAIN / OPERATIONS_PER_RUN;
  
  printf("Will execute %d runs of %d steps each.\n", num_runs, OPERATIONS_PER_RUN);

  // create a timer for the whole run
  unsigned int total_timer = 0;
  CUT_SAFE_CALL(cutCreateTimer(&total_timer));
  
  // compute size of state
  uint32  s_results = NUM_CHAINS * sizeof(uint64);
  
  // allocate and initialize host memory
  uint64* h_results = (uint64*) calloc(1, s_results);
  for(i = 0; i < NUM_CHAINS; i++) {
    h_results[i] = start + i;
  }
    
  // allocate and initialize device memory
  uint64* d_results;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_results, s_results));

  CUT_SAFE_CALL(cutStartTimer(total_timer));
  
  CUDA_SAFE_CALL(hipMemcpy(d_results, h_results, s_results, hipMemcpyHostToDevice));

  double total_run_time = 0.0;

  uint32 run;
  for(run = 0; run < num_runs; run++) {
    unsigned int run_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&run_timer));
    
    uint32 index = OPERATIONS_PER_CHAIN - 1 - run * OPERATIONS_PER_RUN;

#ifdef TEST_INTERMEDIATES
    // print intermediates (for testing against calculate_chains_dump)
    for(i = 0; i < NUM_CHAINS; i++) {
      printf("results[%d] = 0x%16.16llx\n", i, h_results[i]);
    }
#endif
    
    printf("Run %3.3d/%3.3d, starting at index 0x%6.6x... ", run+1, num_runs, index);

    fflush(stdout);
    usleep(500*1000);
    
    CUT_SAFE_CALL(cutStartTimer(run_timer));

#ifdef TEST_INTERMEDIATES    
    CUDA_SAFE_CALL(hipMemcpy(d_results, h_results, s_results, hipMemcpyHostToDevice));
#endif
    
    dim3 gridDims(NUM_BLOCKS, 1, 1);
    dim3 blockDims(NUM_THREADS, 1, 1);
    crunch<<<gridDims, blockDims>>>(d_results, index);
    
    CUDA_SAFE_CALL(hipDeviceSynchronize());

#ifdef TEST_INTERMEDIATES
    CUDA_SAFE_CALL(hipMemcpy(h_results, d_results, s_results, hipMemcpyDeviceToHost));
#endif
    
    CUT_SAFE_CALL(cutStopTimer(run_timer));
    
    float run_time = cutGetTimerValue(run_timer);
    printf("%f ms.\n", run_time);
    total_run_time += run_time;
    fflush(stdout);

    
    CUT_SAFE_CALL(cutDeleteTimer(run_timer));
  }

  CUDA_SAFE_CALL(hipMemcpy(h_results, d_results, s_results, hipMemcpyDeviceToHost));

  CUT_SAFE_CALL(cutStopTimer(total_timer));

  // free device memory
  CUDA_SAFE_CALL(hipFree((void**)d_results));

  // print results
  for(i = 0; i < NUM_CHAINS; i++) {
    printf("results[%d] = 0x%16.16llx\n", i, h_results[i]);
  }

  // free host memory  
  free(h_results);

  // report total time
  printf("Total time: %f ms, %f spent crunching\n", cutGetTimerValue(total_timer), total_run_time);

  // delete the whole-run timer  
  CUT_SAFE_CALL(cutDeleteTimer(total_timer));
  
  return 0;
}
