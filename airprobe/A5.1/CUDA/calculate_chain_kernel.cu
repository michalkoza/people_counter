#include "hip/hip_runtime.h"
/*
 * Calculation of chains for A5/1 rainbow table cracking.
 *
 *
 * Loosely based on: A pedagogical implementation of A5/1.
 *
 * Copyright (C) 1998-1999: Marc Briceno, Ian Goldberg, and David Wagner
 *
 * See accompanying file A5.1.c for original version and full copyright
 *
 *
 * Modified and optimized for running on CUDA.
 *
 * Copyright (C) 2009: Ingo Albrecht <prom@berlin.ccc.de>
 *
 */

// XXX: Undefine for 64bit platform. You will also have to fix up printfs.
#define BITSIZE_32

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Masks for the three shift registers */
#define R1MASK	0x07FFFF /* 19 bits, numbered 0..18 */
#define R2MASK	0x3FFFFF /* 22 bits, numbered 0..21 */
#define R3MASK	0x7FFFFF /* 23 bits, numbered 0..22 */

/* Middle bit of each of the three shift registers, for clock control */
#define R1MID	0x000100 /* bit 8 */
#define R2MID	0x000400 /* bit 10 */
#define R3MID	0x000400 /* bit 10 */

/* Feedback taps, for clocking the shift registers. */
#define R1TAPS	0x072000 /* bits 18,17,16,13 */
#define R2TAPS	0x300000 /* bits 21,20 */
#define R3TAPS	0x700080 /* bits 22,21,20,7 */

/* Output taps, for output generation */
#define R1OUT	0x040000 /* bit 18 (the high bit) */
#define R2OUT	0x200000 /* bit 21 (the high bit) */
#define R3OUT	0x400000 /* bit 22 (the high bit) */

typedef unsigned char byte;
#ifdef BITSIZE_32
typedef unsigned long uint32;
typedef unsigned long long uint64;
#else
typedef unsigned int  uint32;
typedef unsigned long uint64;
#endif

typedef unsigned int bit;

__device__ bit parity32(uint32 x) {
	x ^= x>>16;
	x ^= x>>8;
	x ^= x>>4;
	x ^= x>>2;
	x ^= x>>1;
	return x&1;
}

__device__ uint32 clockone(uint32 reg, uint32 mask, uint32 taps) {
	uint32 t = reg & taps;
	reg = (reg << 1) & mask;
	reg |= parity32(t);
	return reg;
}

__device__ bit majority(uint32 R1, uint32 R2, uint32 R3) {
	int sum;
	sum = ((R1&R1MID) >> 8) + ((R2&R2MID) >> 10) + ((R3&R3MID) >> 10);
	if (sum >= 2)
		return 1;
	else
		return 0;
}

__device__ bit getbit(uint32 R1, uint32 R2, uint32 R3) {
  return ((R1&R1OUT) >> 18) ^ ((R2&R2OUT) >> 21) ^ ((R3&R3OUT) >> 22);
}

__device__ uint64 calculate_link (uint64 input, uint32 count) {
  uint64 result;
  int i;

  /* Reduction function. */
  uint32 R1 = ((input >> (22 + 23))^count) & R1MASK;
  uint32 R2 = ((input >> 23)^count) & R2MASK;
  uint32 R3 = (input^count) & R3MASK;

  result = getbit(R1, R2, R3);
  for(i=1;i<64;i++) {
    // Yes, virginia, we only need to clock 63 times for 64 bits of output

    // clock()
    bit maj = majority(R1, R2, R3);
    uint32 T1 = clockone(R1, R1MASK, R1TAPS);
    uint32 T2 = clockone(R2, R2MASK, R2TAPS);
    uint32 T3 = clockone(R3, R3MASK, R3TAPS);

    if (((R1&R1MID)!=0) == maj)
      R1 = T1;
    if (((R2&R2MID)!=0) == maj)
      R2 = T2;
    if (((R3&R3MID)!=0) == maj)
      R3 = T3;

    result = (result << 1)| getbit(R1, R2, R3);
  }
  return result;
}

__global__ void crunch(uint64* results, uint32 index) {
  uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;

  uint64 state = results[tid];

  uint32 i;
  for(i = 0; i < OPERATIONS_PER_RUN; i++) {
    state = calculate_link(state, index - i);
  }
  
  results[tid] = state;
}

